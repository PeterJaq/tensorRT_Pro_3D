#include "hip/hip_runtime.h"

#include <onnxplugin/onnxplugin.hpp>
#include <hip/hip_runtime.h>
#include <cuda_fp16.hpp>
#include "common/pointpillar_params.h"
#include "common/pointpillars_kernel.h"

using namespace ONNXPlugin;

template <typename T>
void writeToBuffer(char*& buffer, const T& val)
{
    *reinterpret_cast<T*>(buffer) = val;
    buffer += sizeof(T);
}

template <typename T>
T readFromBuffer(const char*& buffer)
{
    T val = *reinterpret_cast<const T*>(buffer);
    buffer += sizeof(T);
    return val;
}

template <typename Element>
__global__ void scatterBEV_kernel(const Element *pillar_features_data,
          const unsigned int *coords_data, const unsigned int *params_data,
          unsigned int featureX, unsigned int featureY,
          Element *spatial_feature_data)
{
    int pillar_idx = blockIdx.x * PILLARS_PER_BLOCK + threadIdx.x;
    int valid_pillars_inBlock = PILLARS_PER_BLOCK;
    const int num_pillars = params_data[0];
    int valid_blocks = (num_pillars+PILLARS_PER_BLOCK-1)/PILLARS_PER_BLOCK;
    if(blockIdx.x >= valid_blocks) return;
    if(blockIdx.x == (valid_blocks-1)) {
      valid_pillars_inBlock = num_pillars % PILLARS_PER_BLOCK;
    }
    valid_pillars_inBlock = (valid_pillars_inBlock==0) ? PILLARS_PER_BLOCK : valid_pillars_inBlock;
    __shared__ Element pillarSM[PILLARS_PER_BLOCK][PILLAR_FEATURE_SIZE]; //pillar*64
    for (int i = 0; i < valid_pillars_inBlock; i++)
    {
      pillarSM[i][threadIdx.x] = pillar_features_data[ (blockIdx.x * PILLARS_PER_BLOCK +i)*PILLAR_FEATURE_SIZE + threadIdx.x];
    }
    __syncthreads();
    if(pillar_idx >= num_pillars) return;
    uint4 coord = ((const uint4 *)coords_data)[pillar_idx];
    unsigned int x = coord.w;
    unsigned int y = coord.z;
    for (int i = 0; i < PILLAR_FEATURE_SIZE; i++)
    {
      spatial_feature_data[i*featureY*featureX + y*featureX + x] = pillarSM[threadIdx.x][i];
    }
}

template <typename Element>
int pillarScatterKernelLaunch(
  int max_pillar_num,
  int num_features,
  const Element *pillar_features_data,
  const unsigned int *coords_data,
  const unsigned int *params_data,
  unsigned int featureX, unsigned int featureY,
  Element *spatial_feature_data,
  hipStream_t stream)
{
    dim3 blocks( (featureX*featureY+PILLARS_PER_BLOCK-1)/PILLARS_PER_BLOCK);
    dim3 threads(PILLARS_PER_BLOCK);
    scatterBEV_kernel<Element><<<blocks, threads, 0, stream>>>(pillar_features_data, coords_data, params_data, featureX, featureY, spatial_feature_data);
    auto err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        return -1;
    }
    return 0;
}

template int pillarScatterKernelLaunch<half>(
  int max_pillar_num,
  int num_features,
  const half *pillar_features_data,
  const unsigned int *coords_data,
  const unsigned int *params_data,
  unsigned int featureX, unsigned int featureY,
  half *spatial_feature_data,
  hipStream_t stream);

template int pillarScatterKernelLaunch<float>(
  int max_pillar_num,
  int num_features,
  const float *pillar_features_data,
  const unsigned int *coords_data,
  const unsigned int *params_data,
  unsigned int featureX, unsigned int featureY,
  float *spatial_feature_data,
  hipStream_t stream);


class PillarScatter : public TRTPlugin {
public:
	SetupPlugin(PillarScatter);

    size_t feature_x_size_;
    size_t feature_y_size_;

    PillarScatter(size_t h, size_t w):feature_y_size_(h), feature_x_size_(w){
    }

    PillarScatter(const void* data, size_t length){
        const char* d = reinterpret_cast<const char*>(data);
        feature_y_size_ = readFromBuffer<size_t>(d);
        feature_x_size_ = readFromBuffer<size_t>(d);
    }

	virtual void config_finish() override{
		 
		// INFO("init hswish config: %s", config_->info_.c_str());
		// INFO("weights = %d", config_->weights_.size());
		// for(int i = 0; i < config_->weights_.size(); ++i){
		// 	auto& w = config_->weights_[i];
		// 	if(w->type() == TRT::DataType::Float16){
		// 		INFO("Weight[%d] shape is %s, dtype = %s, value[0] = %f", i, w->shape_string(), data_type_string(w->type()), float(w->at<__half>(0)));
		// 	}else{
		// 		INFO("Weight[%d] shape is %s, dtype = %s, value[0] = %f", i, w->shape_string(), data_type_string(w->type()), w->at<float>(0));
		// 	}
		// }
	}

	virtual std::shared_ptr<LayerConfig> new_config() override{
		auto cfg = TRTPlugin::new_config();

		//cfg->support_dtype_set_ = {nvinfer1::DataType::kHALF, nvinfer1::DataType::kFLOAT};
		cfg->support_dtype_set_ = {nvinfer1::DataType::kFLOAT};
		return cfg;
	}

	virtual nvinfer1::DimsExprs getOutputDimensions(
        	int32_t outputIndex, const nvinfer1::DimsExprs* inputs, int32_t nbInputs, nvinfer1::IExprBuilder& exprBuilder) noexcept override{
        assert(outputIndex == 0);
        nvinfer1::DimsExprs output;
        output.nbDims = 4;
        output.d[0] = exprBuilder.constant(1);
        output.d[1] = inputs[0].d[1];
        output.d[2] = exprBuilder.constant(feature_y_size_);
        output.d[3] = exprBuilder.constant(feature_x_size_);
        return output;
	}
    
    nvinfer1::DataType getOutputDataType(
    int index, const nvinfer1::DataType* inputTypes, int nbInputs) const noexcept{
        return inputTypes[0];
    }

    bool supportsFormatCombination(
                int32_t pos, const nvinfer1::PluginTensorDesc* inOut, int32_t nbInputs, int32_t nbOutputs) const noexcept{ 
        assert(nbInputs == 3);
        assert(nbOutputs == 1);
        const nvinfer1::PluginTensorDesc& in = inOut[pos];
        if (pos == 0)
        {
            return (in.type == nvinfer1::DataType::kFLOAT || in.type == nvinfer1::DataType::kHALF) && (in.format == nvinfer1::TensorFormat::kLINEAR);
        }
        if (pos == 1)
        {
            return (in.type == nvinfer1::DataType::kINT32) && (in.format == nvinfer1::TensorFormat::kLINEAR);
        }
        if (pos == 2)
        {
            return (in.type == nvinfer1::DataType::kINT32) && (in.format == nvinfer1::TensorFormat::kLINEAR);
        }
        if (pos == 3)
        {
            return (in.type == inOut[0].type) && (in.format == nvinfer1::TensorFormat::kLINEAR || in.format == nvinfer1::TensorFormat::kHWC8);
        }
        return false;
    }

    size_t getSerializationSize() const noexcept{
        return 3 * sizeof(size_t);
    }

    void serialize(void* buffer) const noexcept{
        char* d = reinterpret_cast<char*>(buffer);
        writeToBuffer<size_t>(d, feature_y_size_);
        writeToBuffer<size_t>(d, feature_x_size_);
    }


	virtual int32_t enqueue(const nvinfer1::PluginTensorDesc* inputDesc, const nvinfer1::PluginTensorDesc* outputDesc, 
            const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept override{
        try
        {
            int maxPillarNum = inputDesc[0].dims.d[0];
            int numFeatures = inputDesc[0].dims.d[1];
            
            nvinfer1::DataType inputType = inputDesc[0].type;

            auto coords_data = static_cast<const unsigned int *>(inputs[1]);
            auto params_data = static_cast<const unsigned int *>(inputs[2]);

            unsigned int featureY = feature_y_size_;
            unsigned int featureX = feature_x_size_;

            int status = -1;

            if(inputType == nvinfer1::DataType::kHALF){
                auto pillar_features_data = static_cast<const half *>(inputs[0]);
                auto spatial_feature_data = static_cast<half *>(outputs[0]);
                hipMemsetAsync(spatial_feature_data, 0, numFeatures*featureY*featureX * sizeof(half), stream);
                status = pillarScatterKernelLaunch<half>(
                    maxPillarNum,
                    numFeatures,
                    pillar_features_data,
                    coords_data,
                    params_data,
                    featureX,
                    featureY,
                    spatial_feature_data,
                    stream
                    );
                assert(status == STATUS_SUCCESS);
                return status;
            }
            else if(inputType == nvinfer1::DataType::kFLOAT){
                auto pillar_features_data = static_cast<const float *>(inputs[0]);
                auto spatial_feature_data = static_cast<float *>(outputs[0]);
                hipMemsetAsync(spatial_feature_data, 0, numFeatures*featureY*featureX * sizeof(float), stream);
                status = pillarScatterKernelLaunch<float>(
                    maxPillarNum,
                    numFeatures,
                    pillar_features_data,
                    coords_data,
                    params_data,
                    featureX,
                    featureY,
                    spatial_feature_data,
                    stream
                    );
                assert(status == STATUS_SUCCESS);
                return status;
            }
            else{
                assert(status == STATUS_SUCCESS);
                return status;
            }
        }
        catch (const std::exception& e)
        {
            std::cerr << e.what() << std::endl;
        }
        return -1;
    }

    int enqueue(const std::vector<GTensor>& inputs, std::vector<GTensor>& outputs, const std::vector<GTensor>& weights, void* workspace, hipStream_t stream) override{
		return 0;
	}
};
// RegisterPillarScatterPlugin(PillarScatter);

class PillarScatterPluginCreator : public nvinfer1::IPluginCreator{																				
	public:																																			
		PillarScatterPluginCreator(){																																	
			mPluginAttributes.clear();																												
			mPluginAttributes.emplace_back(nvinfer1::PluginField("dense_shape", nullptr, nvinfer1::PluginFieldType::kINT32, 1));					
			mFC.nbFields = mPluginAttributes.size();																										
			mFC.fields = mPluginAttributes.data();																									
		}																																			
		const char* getPluginName() const noexcept override{return "PillarScatter";}																		
		const char* getPluginVersion() const noexcept override{return "1";}																			
		const nvinfer1::PluginFieldCollection* getFieldNames() noexcept override{return &mFieldCollection;}											
																																						
		nvinfer1::IPluginV2DynamicExt* createPlugin(const char* name, const nvinfer1::PluginFieldCollection* fc) noexcept override{					
			mPluginName = name;																														
			const nvinfer1::PluginField* fields = fc->fields;																						
			int nbFields = fc->nbFields;																											
			int target_h = 0;																														
			int target_w = 0;																														
			for (int i = 0; i < nbFields; ++i)																										
			{																																		
				const char* attr_name = fields[i].name;																								
				if (!strcmp(attr_name, "dense_shape"))																								
				{																																	
					const int* ts = static_cast<const int*>(fields[i].data);																		
					target_h = ts[0];																												
					target_w = ts[1];																												
				}																																	
			}																																		
			auto* plugin = new PillarScatter(																												
				target_h,																															
				target_w																															
			);																																		
			return plugin;																															
		}																																			
																																					
		nvinfer1::IPluginV2DynamicExt* deserializePlugin(const char* name, const void* serialData, size_t serialLength) noexcept override{			
		auto plugin = new PillarScatter(serialData, serialLength);																							
			plugin->pluginInit(name, serialData, serialLength);																						
			mPluginName = name;																														
			return plugin;																															
		}																																			
		void setPluginNamespace(const char* libNamespace) noexcept override{mNamespace = libNamespace;}												
		const char* getPluginNamespace() const noexcept override{return mNamespace.c_str();}														
																																					
	private:																																		
		std::string mNamespace;																														
		std::string mPluginName;																													
    	static nvinfer1::PluginFieldCollection mFC;																									
		static std::vector<nvinfer1::PluginField> mPluginAttributes;																														\
		nvinfer1::PluginFieldCollection mFieldCollection{0, nullptr};																				
	};																																				
	REGISTER_TENSORRT_PLUGIN(PillarScatterPluginCreator);